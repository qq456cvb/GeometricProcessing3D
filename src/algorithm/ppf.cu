#include "hip/hip_runtime.h"
#include "algorithm/ppf.h"
#include <hip/hip_runtime.h>
#include "utils/hip/hip_vector_types.h"
#include <armadillo>

#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/binary_search.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/zip_iterator.h>


inline __device__ uint32_t rotl32(uint32_t x, int8_t r) {
  return (x << r) | (x >> (32 - r));
}


inline __device__ float angle_between(const float3 &a, const float3 &b) {
    return atan2(length(cross(a, b)), dot(a, b));
}


inline __device__ uint32_t murmurppf(const uint32_t ppf[4]) {
    uint32_t h1 = 42;
    const uint32_t c1 = 0xcc9e2d51;
    const uint32_t c2 = 0x1b873593;

    for (int i = 0; i < 4; i++) {
        uint32_t k1 = ppf[i];

        k1 *= c1;
        k1 = rotl32(k1, 15);
        k1 *= c2;

        h1 ^= k1;
        h1 = rotl32(h1, 13); 
        h1 = h1 * 5 + 0xe6546b64;
    }

    h1 ^= 16;
    h1 ^= h1 >> 16;
    h1 *= 0x85ebca6b;
    h1 ^= h1 >> 13;
    h1 *= 0xc2b2ae35;
    h1 ^= h1 >> 16;

    return h1;
}


__device__ uint32_t compute_ppfhash (
    const float3& p1, const float3& n1,
    const float3& p2, const float3& n2,
    const float dist_delta, const float angle_delta) {

    // Compute the vector between the points
    float3 d = make_float3(p2.x - p1.x, p2.y - p1.y, p2.z - p1.z);

    // Compute the 4 components of the ppf feature
    const float f1 = length(d);
    const float f2 = angle_between(d, n1);
    const float f3 = angle_between(d, n2);
    const float f4 = angle_between(n1, n2);

    // Discretize the PPF Feature before hashing
    uint32_t feature[4];
    feature[0] = static_cast<uint32_t>(f1 / dist_delta);
    feature[1] = static_cast<uint32_t>(f2 / angle_delta);
    feature[2] = static_cast<uint32_t>(f3 / angle_delta);
    feature[3] = static_cast<uint32_t>(f4 / angle_delta);

    // Return the hash of the feature.
    return murmurppf(feature);
}


struct PPFKernel
{
    const float3 *pc, *pc_normal, *transforms;
    uint64_t *ppf_codes;
    int npoints;
    float dist_delta, angle_delta;

    PPFKernel(const thrust::device_vector<float3> &pc, 
            const thrust::device_vector<float3> &pc_normal, 
            const thrust::device_vector<float> &transforms, 
            thrust::device_vector<uint64_t> &ppf_codes, 
            const int npoints,
            const float dist_delta,
            const float angle_delta) : 
        pc(thrust::raw_pointer_cast(&pc[0])), 
        pc_normal(thrust::raw_pointer_cast(&pc_normal[0])),
        transforms((float3 *)thrust::raw_pointer_cast(&transforms[0])),
        ppf_codes(thrust::raw_pointer_cast(&ppf_codes[0])),
        npoints(npoints),
        dist_delta(dist_delta),
        angle_delta(angle_delta) {}

    __device__ void operator()(const uint64_t &n) const {
        int a = n / npoints;
        int b = n % npoints;
        const float3 *t = &transforms[a * 3];  // transform point a to axis x
        uint32_t ppf_hash = compute_ppfhash(pc[a], pc_normal[a], pc[b], pc_normal[b], dist_delta, angle_delta);
        float angle = atan2(-dot(t[2], pc[b]), dot(t[1], pc[b])); // in PPF paper, left-handed coordinates
        uint64_t angle_bin = static_cast<uint64_t>(angle / angle_delta);
        uint64_t code = (static_cast<uint64_t>(ppf_hash) << 32) | 
                (static_cast<uint64_t>(a) << 6) | 
                angle_bin;

        // Save the code
        ppf_codes[n] = code;
    }
};


struct TransXKernel
{
    const float3 *pc_normal;
    float *transforms;
    TransXKernel(const thrust::device_vector<float3> &pc_normal,
            thrust::device_vector<float> &transforms) :
        pc_normal(thrust::raw_pointer_cast(&pc_normal[0])),
        transforms(thrust::raw_pointer_cast(&transforms[0])) {}

    __device__ void operator()(int i) const {
        const auto &n = pc_normal[i];
        auto y2 = n.y * n.y;
        auto z2 = n.z * n.z;
        auto yz = n.y * n.z;
        auto y2z2 = y2 + z2;

        // TODO: what if y2z2 == 0
        transforms[i * 9] = n.x;
        transforms[i * 9 + 1] = n.y;
        transforms[i * 9 + 2] = n.z;
        transforms[i * 9 + 3] = -n.y;
        transforms[i * 9 + 4] = 1 + (n.x - 1) * y2 / y2z2;
        transforms[i * 9 + 5] = (n.x - 1) * yz / y2z2;
        transforms[i * 9 + 6] = -n.z;
        transforms[i * 9 + 7] = transforms[i * 9 + 5];
        transforms[i * 9 + 8] = 1 + (n.x - 1) * z2 / y2z2;
    }
};



PPF::PPF(const float &dist_delta, const float &angle_delta) :
    dist_delta(dist_delta),
    angle_delta(angle_delta)
{
}

PPF::~PPF()
{
}


void PPF::setup_model(const PointCloud &model) {
    model_pc = thrust::device_vector<float3>(reinterpret_cast<const float3*>(&(*model.verts.begin())), reinterpret_cast<const float3*>(&(*model.verts.end())));
    model_pc_normal = thrust::device_vector<float3>(reinterpret_cast<const float3*>(&(*model.normals.begin())), reinterpret_cast<const float3*>(&(*model.normals.end())));
    // float3 *pc_ptr = thrust::raw_pointer_cast(model_pc.data());
    // float3 *pc_normal_ptr = thrust::raw_pointer_cast(model_pc_normal.data());
    int npoints = static_cast<int>(model_pc.size());

    model_transforms.resize(npoints * 9);
    // float *transforms_ptr = thrust::raw_pointer_cast(transforms.data());
    TransXKernel transx_kern(model_pc_normal, model_transforms);
    thrust::for_each_n(thrust::counting_iterator<size_t>(0), model_pc_normal.size(), transx_kern);
    // thrust::for_each_n(thrust::counting_iterator<size_t>(0), pc_normal.size(), [=] __device__ (int i) {
    //     const auto &n = pc_normal_ptr[i];
    //     auto y2 = n.y * n.y;
    //     auto z2 = n.z * n.z;
    //     auto yz = n.y * n.z;
    //     auto y2z2 = y2 + z2;

    //     // TODO: what if y2z2 == 0
    //     transforms_ptr[i * 9] = n.x;
    //     transforms_ptr[i * 9 + 1] = n.y;
    //     transforms_ptr[i * 9 + 2] = n.z;
    //     transforms_ptr[i * 9 + 3] = -n.y;
    //     transforms_ptr[i * 9 + 4] = 1 + (n.x - 1) * y2 / y2z2;
    //     transforms_ptr[i * 9 + 5] = (n.x - 1) * yz / y2z2;
    //     transforms_ptr[i * 9 + 6] = -n.z;
    //     transforms_ptr[i * 9 + 7] = transforms_ptr[i * 9 + 5];
    //     transforms_ptr[i * 9 + 8] = 1 + (n.x - 1) * z2 / y2z2;
    // });

    // thrust::host_vector<float> h_transforms(transforms);
    // for (size_t i = 0; i < 10; i++) {
    //     auto v = arma::fvec(const_cast<float *>(model.normals[i].data()), 3, false, true);
    //     auto R = arma::fmat(const_cast<float *>(&h_transforms[i * 9]), 3, 3, false, true);
    //     std::cout << R.t() * v << std::endl;
    // }

    model_ppf_codes.resize(model.verts.size() * model.verts.size());
    PPFKernel ppf_kern(model_pc, model_pc_normal, model_transforms, model_ppf_codes, npoints, dist_delta, angle_delta);
    thrust::for_each_n(thrust::counting_iterator<size_t>(0), model_ppf_codes.size(), ppf_kern);

    thrust::sort(model_ppf_codes.begin(), model_ppf_codes.end());

    key2ppf.resize(model_ppf_codes.size());
    thrust::device_vector<uint32_t> hash_keys{model_ppf_codes.size(), 0};
    uint32_t *key2ppf_ptr = thrust::raw_pointer_cast(key2ppf.data());
    uint32_t *hash_keys_ptr = thrust::raw_pointer_cast(hash_keys.data());
    uint64_t *ppf_codes_ptr = thrust::raw_pointer_cast(model_ppf_codes.data());
    thrust::for_each_n(thrust::counting_iterator<size_t>(0), model_ppf_codes.size(), [=] __device__ (int i) {
        key2ppf_ptr[i] = static_cast<uint32_t>(0xffffffc0 & ppf_codes_ptr[i]);
        hash_keys_ptr[i] = static_cast<uint32_t>(ppf_codes_ptr[i] >> 32);
    });

    thrust::equal_to<uint32_t> binary_pred;
    thrust::plus<uint32_t> binary_op;
    ppf_count.resize(hash_keys.size());
    unique_hash_keys.resize(hash_keys.size());
    // printf("size before: %ld\n", hash_keys.size());
    // printf("%u\n", thrust::reduce(ppf_count.begin(), ppf_count.end(), 0, thrust::maximum<uint32_t>()));
    auto end = thrust::reduce_by_key(hash_keys.begin(), hash_keys.end(), thrust::make_constant_iterator(1), unique_hash_keys.begin(), ppf_count.begin(), binary_pred, binary_op);
    unique_hash_keys.resize(thrust::distance(unique_hash_keys.begin(), end.first));
    ppf_count.resize(thrust::distance(ppf_count.begin(), end.second));
    // printf("size after: %ld\n", unique_hash_keys.size());
    // printf("%u\n", thrust::reduce(ppf_count.begin(), ppf_count.end(), 0, thrust::maximum<uint32_t>()));

    first_ppf_idx.resize(ppf_count.size());
    thrust::exclusive_scan(ppf_count.begin(), ppf_count.end(), first_ppf_idx.begin());
}


void PPF::detect(const PointCloud &scene) {

}